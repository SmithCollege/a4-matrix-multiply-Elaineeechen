
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define SIZE 10
#define TILE_WIDTH 32

double get_clock() {
  struct timeval tv; int ok;
  ok = gettimeofday(&tv, (void *) 0);
  if (ok<0) { printf("gettimeofday error"); }
  return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

__global__ void MatrixMulKernel(float* d_M, float* d_N, float* d_P, int Width) {
  int Row = blockIdx.y*blockDim.y+threadIdx.y;
  int Col = blockIdx.x*blockDim.x+threadIdx.x;
  if ((Row < Width) && (Col < Width)) {
     float Pvalue = 0;
     for (int k = 0; k < Width; ++k){
     	 Pvalue += d_M[Row * Width + k] * d_N[k * Width + Col];
     }
     d_P[Row * Width + Col] = Pvalue;
  }
}

int main() {
  float* M = (float*)malloc(sizeof(float) * SIZE * SIZE);
  float* N = (float*)malloc(sizeof(float) * SIZE * SIZE);
  float* P = (float*)malloc(sizeof(float) * SIZE * SIZE);
  float *d_M, *d_N, *d_P;
  hipMalloc(&d_M, SIZE* SIZE * sizeof(float));
  hipMalloc(&d_N, SIZE* SIZE * sizeof(float));
  hipMalloc(&d_P, SIZE* SIZE * sizeof(float));

  for (int i = 0; i < SIZE; i++) {
    for (int j = 0; j < SIZE; j++) {
      M[i*SIZE+j] = 1;
      N[i*SIZE+j] = 1;
    }
  }

  hipMemcpy(d_M, M, SIZE * SIZE * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_N, N, SIZE * SIZE * sizeof(float), hipMemcpyHostToDevice);

  dim3 dimGrid(ceil((1.0*SIZE)/TILE_WIDTH), ceil((1.0*SIZE)/TILE_WIDTH), 1);
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

  double t0 = get_clock();
  MatrixMulKernel<<<dimGrid, dimBlock>>>(d_M, d_N, d_P, SIZE);
  hipDeviceSynchronize();
  double t1 = get_clock();

  hipMemcpy(P, d_P, SIZE * SIZE *sizeof(float), hipMemcpyDeviceToHost);

  for (int i = 0; i < SIZE; i++) {
    for (int j = 0; j < SIZE; j++) {
      if (P[i*SIZE+j] != SIZE) {
        printf("Error at z[%d][%d]: %f\n", i, j, P[i*SIZE+j]);
      }
    }
  }
  printf("Time: %f ns\n", 1000000000.0*(t1 - t0));

  return 0;    
}