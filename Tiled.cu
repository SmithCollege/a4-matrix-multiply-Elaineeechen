
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define SIZE 10
#define TILE_WIDTH 32

double get_clock() {
  struct timeval tv; int ok;
  ok = gettimeofday(&tv, (void *) 0);
  if (ok<0) { printf("gettimeofday error"); }
  return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

__global__ void MatrixMulKernel(float* d_M, float* d_N, float* d_P, int Width) {
  __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx;
  float Pvalue = 0;
  for (int m = 0; m < (Width + TILE_WIDTH - 1) / TILE_WIDTH; ++m) {
      if ((Row < Width) && (m*TILE_WIDTH + tx < Width)) {
      	 Mds[ty][tx] = d_M[Row*Width + m*TILE_WIDTH + tx];
      } else {
      	Mds[ty][tx] = 0;
      }
      if ((Col < Width) && (m*TILE_WIDTH + ty < Width)) {
      	 Nds[ty][tx] = d_N[(m*TILE_WIDTH + ty)*Width + Col];
      } else {
      	Nds[ty][tx] = 0;
      }
      __syncthreads();
      for (int k = 0; k < TILE_WIDTH; ++k) {
      	  Pvalue += Mds[ty][k] * Nds[k][tx];
      }
      __syncthreads();
  }
  if ((Row < Width) && (Col < Width)) {
     d_P[Row*Width + Col] = Pvalue;
  }
}

int main() {
  float* M = (float*)malloc(sizeof(float) * SIZE * SIZE);
  float* N = (float*)malloc(sizeof(float) * SIZE * SIZE);
  float* P = (float*)malloc(sizeof(float) * SIZE * SIZE);
  float *d_M, *d_N, *d_P;
  hipMalloc(&d_M, SIZE* SIZE * sizeof(float));
  hipMalloc(&d_N, SIZE* SIZE * sizeof(float));
  hipMalloc(&d_P, SIZE* SIZE * sizeof(float));

  for (int i = 0; i < SIZE; i++) {
    for (int j = 0; j < SIZE; j++) {
      M[i*SIZE+j] = 1;
      N[i*SIZE+j] = 1;
    }
  }

  hipMemcpy(d_M, M, SIZE * SIZE * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_N, N, SIZE * SIZE * sizeof(float), hipMemcpyHostToDevice);

  dim3 dimGrid(ceil((1.0*SIZE)/TILE_WIDTH), ceil((1.0*SIZE)/TILE_WIDTH), 1);
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

  double t0 = get_clock();
  MatrixMulKernel<<<dimGrid, dimBlock>>>(d_M, d_N, d_P, SIZE);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Kernel launch failed: %s\n", hipGetErrorString(err));
    return EXIT_FAILURE;
  }
  double t1 = get_clock();

  hipMemcpy(P, d_P, SIZE * SIZE *sizeof(float), hipMemcpyDeviceToHost);

  for (int i = 0; i < SIZE; i++) {
    for (int j = 0; j < SIZE; j++) {
      if (P[i*SIZE+j] != SIZE) {
        printf("Error at z[%d][%d]: %f\n", i, j, P[i*SIZE+j]);
      }
    }
  }
  printf("Time: %f ns\n", 1000000000.0*(t1 - t0));

  return 0;
}
