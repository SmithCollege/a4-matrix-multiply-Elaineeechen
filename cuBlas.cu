#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>
#include <sys/time.h>

#define MATRIX_SIZE 10

double get_clock() {
  struct timeval tv; int ok;
  ok = gettimeofday(&tv, (void *) 0);
  if (ok<0) { printf("gettimeofday error"); }
  return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

void init(float *data, int size) {
    for (int i = 0; i < size; ++i) {
        data[i] = 1;
    }
}

// Perform matrix multiplication on the GPU using cuBLAS
void matrixMultiplyCUDA() {
    const int size = MATRIX_SIZE * MATRIX_SIZE;
    const float alpha = 1.0f;
    const float beta = 0.0f;

    float *h_A = (float *)malloc(size * sizeof(float));
    float *h_B = (float *)malloc(size * sizeof(float));
    float *h_C = (float *)malloc(size * sizeof(float));

    init(h_A, size);
    init(h_B, size);

    float *d_A, *d_B, *d_C;
    hipMalloc((float **)&d_A, size * sizeof(float));
    hipMalloc((float **)&d_B, size * sizeof(float));
    hipMalloc((float **)&d_C, size * sizeof(float));

    hipMemcpy(d_A, h_A, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size * sizeof(float), hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    double t0 = get_clock();
    // Matrix multiplication: C = A * B using cuBLAS (column-major order)
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, MATRIX_SIZE, MATRIX_SIZE, MATRIX_SIZE, &alpha, d_B, MATRIX_SIZE, d_A, MATRIX_SIZE, &beta, d_C, MATRIX_SIZE);
    double t1 = get_clock();

    // Copy result back to host
    hipMemcpy(h_C, d_C, size * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < MATRIX_SIZE; i++) {
    	for (int j = 0; j < MATRIX_SIZE; j++) {
      	    if (h_C[i*MATRIX_SIZE+j] != MATRIX_SIZE) {
               printf("Error at c[%d][%d]: %f\n", i, j, h_C[i*MATRIX_SIZE+j]);
      	    }
    	}
    }

    printf("Time: %f ns\n", 1000000000.0*(t1 - t0));

    // Clean up
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
}

int main() {
    matrixMultiplyCUDA();
    return 0;
}


